#include "hip/hip_runtime.h"
//AUTHOR: Prabhav Talukdar
//        EE23M053
//Tiled Matrix multiplication on using cuda accelerated algorithm

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<stdlib.h>

#define N 64
#define BlockSize 4


__global__ void mat_gpu(int *A, int *B, int *C){
    __shared__ int shared_A[N*N];
    __shared__ int shared_B[N*N];

    unsigned bx= blockIdx.x;
    unsigned by= blockIdx.y;
    unsigned tx= threadIdx.x;
    unsigned ty= threadIdx.y;
    
    unsigned row= by * blockDim.x + ty;
    unsigned col= bx * blockDim.y + tx;

    int acc=0;
    // Bringing block/tile from global memory to shared memory.
    for (unsigned i=0; i < (N/BlockSize); i++){
        //  Data is stored in row major format. A = [ row1[N] row2[N] row3[N] .. ]
        // [ row * N + ( i * BlockSize + tx )]:
        //         Here row * N traverses through every row stored in A. Its the zeroth position
        //         i * BlockSize determines the block which we are operating and tx points the element
        // [ (i * BlockSize + ty ) * N + col ]:
        //         col represents the zeroth position
        //         (i*BlockSize)*N is the block below present block
        //         ty*N is traversing through the colomn  
                      
        shared_A[ty*BlockSize+tx]= A[ row * N + ( i * BlockSize + tx )];
        shared_B[ty*BlockSize+tx]= B[ (i * BlockSize)*N + (ty*N) + col];
        __syncthreads();
        
        for(unsigned j=0; j< BlockSize; j++){
            acc += shared_A[ty*BlockSize+j] * shared_B[j*BlockSize+tx]; 
        }
        __syncthreads();
    }
    C[row*N+col]= acc;
}

int matMul_verify(int *A, int *B, int *C){
    int ans[N*N]={0};
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            for(int k=0; k<N; k++){
                ans[i*N+j]+=A[i*N+k]*B[k*N+j];
            }
        }
    }
    for(int i=0;i<N*N;i++){
        if(ans[i]!=C[i]){
            return 1;
        }
    }
    return 0;
}

int main()
{
    int *a,*b,*c;
    int *gpu_a, *gpu_b, *gpu_c;

    a =(int*)malloc(N*N*sizeof(int));
    b =(int*)malloc(N*N*sizeof(int));
    c =(int*)malloc(N*N*sizeof(int));
        
    hipMalloc(&gpu_a, N*N*sizeof(int));
    hipMalloc(&gpu_b, N*N*sizeof(int));
    hipMalloc(&gpu_c, N*N*sizeof(int));
    
    for(int i=0;i<N*N;i++){
        a[i]=i%10; //rand()%10;
        b[i]=i%10; //rand()%10;
        c[i]=-1;
    }
    hipMemcpy(gpu_a, a, N*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, N*N*sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy(gpu_c, c, N*N*sizeof(int), hipMemcpyHostToDevice);

    int gridSize= (int)ceil(N/BlockSize);
    int THREADS = BlockSize;

    dim3 threads(THREADS, THREADS, 1);
    dim3 grid(gridSize, gridSize, 1);
    
    
    mat_gpu<<<grid,threads>>>(gpu_a, gpu_b, gpu_c);

    hipMemcpy(c, gpu_c, N*N*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    clock_t cpu_startTime, cpu_endTime;
    long double cpu_ElapseTime=0;
    cpu_startTime = clock();

    matMul_verify(a,b,c);
    
    cpu_endTime = clock();

    cpu_ElapseTime = ((cpu_endTime - cpu_startTime)/CLOCKS_PER_SEC)*1000000;
    
    int flag=0;

    if(flag==1){
        printf("Wrong Calculation\n");
    }
    else{
        printf("Correct\n");
    }

    // printf("Input Matrix %dx%d",N,N);
    // for(int i=0; i< N*N; i++){
    //         if(i%N==0){
    //             printf("\n");
    //         }
    //         printf("%d ", a[i]);
    // }
    // printf("\n");

    // for(int i=0; i< N*N; i++){
    //         if(i%N==0){
    //             printf("\n");
    //         }
    //         printf("%d ", b[i]);
    // }
    // printf("\n");
    // printf("Output Matrix");
    // for(int i=0; i< N*N; i++){
    //         if(i%N==0){
    //             printf("\n");
    //         }
    //         printf("%d ", c[i]);
    // }
    printf("CPU Calculation Elapsed Time: %Lf", cpu_ElapseTime);
    free(a);
    free(b);
    free(c);
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    return 0;
}